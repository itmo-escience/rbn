#include "hip/hip_runtime.h"
// Compiled with CUDA C++

#include "dev_find_attractor.hpp"

#include <iostream>
#include <thrust/functional.h>
#include <thrust/logical.h>
#include <thrust/fill.h>
#include <thrust/device_vector.h>
#include <thrust/swap.h>

#include "boolean_functions.hpp"

namespace gpu_acc {

namespace {

struct dev_rm {
	~dev_rm() {
		hipDeviceReset();
	}
};

dev_rm& dev_init() {
	static dev_rm drm;
	return drm;
}

__global__
void update_state(int iter, int nodes_count, const int* xs, int* ys, node_behavior* behavior,
					boolean_functions fs, int* ref, bool* eq_to_ref)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if(idx < nodes_count) {
		int x = xs[idx];
		int y = fs(idx, xs);
		behavior[idx].changes += (x != y);
		behavior[idx].sum += y;
		ys[idx] = y;
		if(y != ref[idx]) {
			*eq_to_ref = false;
		}
	}
}
/*
__device__ unsigned int threads_working;

__device__
void sync_system() {
	while(threads_working != 0);
}

__global__
void find_attractor_kernel(int nodes_count, const int* xs, int* ys, node_behavior* behavior,
							boolean_functions fs, int* reference_state) {
	threads_working = 0;
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if(idx == 0) {
		//
	}
	sync_system();

	for(int i = 0; i < 100000; ++i) {
		update_state(idx, nodes_count, xs, ys, behavior[idx], fs, reference_state[idx]);
		sync_system();
	}
}*/

} // namespace

int dev_find_attractor(network& net) {
	dev_init();
	unsigned int T[] = {100, 1000, 10000, 100000};
	const int max = sizeof(T) / sizeof(unsigned int) - 1;
	unsigned int i, k;
	size_t nodes_count = net.state().size();
	size_t threads_count = 512;
	size_t blocks_count = nodes_count / threads_count + 1;
	if(nodes_count % threads_count == 0) {
		--blocks_count;
	}
	
	thrust::device_vector<int> state0 = net.state(), xs = net.state(), ys = net.state();
	thrust::device_vector<node_behavior> dev_behavior = net.behavior();
	thrust::device_vector<int> functions_storage = net.functions();
	boolean_functions bfs(net.function_size(), net.arguments_size(), functions_storage);
	
	int attractor_found_on = 0;
	// Synchronizing CPU and GPU after each kernel invokation (update_state) is very expensive
	// So we run (check_state_each) iterations, and then check if attractor was found
	int check_state_each = 98;
	thrust::device_vector<bool> dev_eq_to_ref(check_state_each);
	thrust::fill(dev_eq_to_ref.begin(), dev_eq_to_ref.end(), true);
	for(i = 1, k = 0; i < T[max]; ++i){
		int* xs_ptr = thrust::raw_pointer_cast(xs.data());
		int* ys_ptr = thrust::raw_pointer_cast(ys.data());
		node_behavior* b_ptr = thrust::raw_pointer_cast(dev_behavior.data());
		int* ref_ptr = thrust::raw_pointer_cast(state0.data());
		bool* eq_to_ref_ptr = thrust::raw_pointer_cast(dev_eq_to_ref.data());
		
		update_state<<<blocks_count, threads_count>>> (i,
			nodes_count,
			xs_ptr,
			ys_ptr,
			b_ptr,
			bfs,
			ref_ptr, // reference state
			eq_to_ref_ptr + ((i - 1) % check_state_each)
		);

		xs.swap(ys);

		if(i % check_state_each == 0) {
			thrust::device_vector<bool>::iterator eq_to_ref_on = thrust::find(dev_eq_to_ref.begin(), dev_eq_to_ref.end(), true);
			if(eq_to_ref_on != dev_eq_to_ref.end()) {
				attractor_found_on = ((i - 1) / check_state_each) * check_state_each + thrust::distance(dev_eq_to_ref.begin(), eq_to_ref_on) + 1;
			}
			thrust::fill(dev_eq_to_ref.begin(), dev_eq_to_ref.end(), true);
		}
		if(attractor_found_on > 0) {
			break;
		}

		if(i == T[k]){
			++k;
			//cout << "ts";
			state0 = xs;
			thrust::fill(dev_behavior.begin(), dev_behavior.end(), node_behavior());
		}
	}
	
	thrust::host_vector<int> state1 = xs;
	net.state() = std::vector<int>(state1.begin(), state1.end());
	
	thrust::host_vector<node_behavior> behavior = dev_behavior;
	net.behavior() = std::vector<node_behavior>(behavior.begin(), behavior.end());
	
	if(attractor_found_on == 0){
		std::cout << "koniec";// << std::endl;
		return T[max] - T[max - 1];
	}
	else if(k > 0) {
		return attractor_found_on - T[k-1];
	}
	else return attractor_found_on;
}
	
} // namespace gpu_acc
