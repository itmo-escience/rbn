#include "hip/hip_runtime.h"
// Compiled with CUDA C++

#include "dev_find_attractor.hpp"

#include <iostream>
#include <thrust/functional.h>
#include <thrust/logical.h>
#include <thrust/fill.h>
#include <thrust/device_vector.h>
#include <thrust/swap.h>

#include "boolean_functions.hpp"

namespace gpu_acc {

namespace {

__global__
void update_state(int nodes_count, const int* xs, int* ys, node_behavior* behavior, boolean_functions fs, int* reference_state, int* eq_to_ref) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if(i < nodes_count) {
		int x = xs[i];
		int y = fs(i, xs);
		behavior[i].changes += (x != y);
		behavior[i].sum += y;
		ys[i] = y;
		eq_to_ref[i] = (y == reference_state[i]);
	}
}

} // namespace

struct dev_rm {
	~dev_rm() {
		hipDeviceReset();
	}
};

dev_rm& dev_init() {
	static dev_rm drm;
	return drm;
}

int dev_find_attractor(network& net) {
	dev_init();
	unsigned int T[] = {100, 1000, 10000, 100000};
	const int max = sizeof(T) / sizeof(unsigned int) - 1;
	unsigned int i, k;
	size_t nodes_count = net.state().size();
	size_t blocks_count = nodes_count / 32 + 1;
	if(nodes_count % 32 == 0) {
		--blocks_count;
	}
	size_t threads_count = 32;
	
	thrust::device_vector<int> state0 = net.state(), xs = net.state(), ys = net.state();
	thrust::device_vector<node_behavior> dev_behavior = net.behavior();
	thrust::device_vector<int> functions_storage = net.functions();
	boolean_functions bfs(net.function_size(), net.arguments_size(), functions_storage);
	thrust::device_vector<int> equal_to_reference_state(nodes_count);

	for(i = 1, k = 0; i < 100000/*T[max]*/; ++i){
		int* xs_ptr = thrust::raw_pointer_cast(xs.data());
		int* ys_ptr = thrust::raw_pointer_cast(ys.data());
		node_behavior* b_ptr = thrust::raw_pointer_cast(dev_behavior.data());
		int* ref_ptr = thrust::raw_pointer_cast(state0.data());
		int* ref_eq_ptr = thrust::raw_pointer_cast(equal_to_reference_state.data());

		update_state<<<blocks_count, threads_count>>> (
			nodes_count,
			xs_ptr,
			ys_ptr,
			b_ptr,
			bfs,
			ref_ptr, // reference state
			ref_eq_ptr
		);

		xs.swap(ys);

		//std::cout << i << ": " << reduction_result  << std::endl;
		if(thrust::any_of(equal_to_reference_state.begin(), equal_to_reference_state.end(), thrust::identity<bool>())) {
			//break;
		}

		if(i == T[k]){
			++k;
			//cout << "ts";
			state0 = xs;
			thrust::fill(dev_behavior.begin(), dev_behavior.end(), node_behavior());
		}
	}
	
	thrust::host_vector<int> state1 = xs;
	net.state() = std::vector<int>(state1.begin(), state1.end());
	
	thrust::host_vector<node_behavior> behavior = dev_behavior;
	net.behavior() = std::vector<node_behavior>(behavior.begin(), behavior.end());
	
	if(i == T[max]){
		std::cout << "koniec";// << std::endl;
		return i - T[max - 1];
	}
	else if(k > 0) {
		return i - T[k-1];
	}
	else return i;
}
	
} // namespace gpu_acc
