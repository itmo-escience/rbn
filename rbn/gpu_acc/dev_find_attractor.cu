#include "hip/hip_runtime.h"
// Compiled with CUDA C++

#include "dev_find_attractor.hpp"

#include <iostream>
#include <thrust/functional.h>
#include <thrust/logical.h>
#include <thrust/fill.h>
#include <thrust/device_vector.h>
#include <thrust/swap.h>

#include "boolean_functions.hpp"

namespace gpu_acc {

namespace {

struct dev_rm {
	~dev_rm() {
		hipDeviceReset();
	}
};

dev_rm& dev_init() {
	static dev_rm drm;
	return drm;
}


__device__ bool dev_eq_to_ref;

__global__
void update_state(int nodes_count, const int* xs, int* ys, node_behavior* behavior,
					boolean_functions fs, int* ref)//, bool* eq_to_ref)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if(idx < nodes_count) {
		int x = xs[idx];
		int y = fs(idx, xs);
		behavior[idx].changes += (x != y);
		behavior[idx].sum += y;
		ys[idx] = y;
		if(y != ref[idx]) {
			dev_eq_to_ref = false;
		}
	}
}
/*
__device__ unsigned int threads_working;

__device__
void sync_system() {
	while(threads_working != 0);
}

__global__
void find_attractor_kernel(int nodes_count, const int* xs, int* ys, node_behavior* behavior,
							boolean_functions fs, int* reference_state) {
	threads_working = 0;
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if(idx == 0) {
		//
	}
	sync_system();

	for(int i = 0; i < 100000; ++i) {
		update_state(idx, nodes_count, xs, ys, behavior[idx], fs, reference_state[idx]);
		sync_system();
	}
}*/

} // namespace

int dev_find_attractor(network& net) {
	dev_init();
	unsigned int T[] = {100, 1000, 10000, 1000000};
	const int max = sizeof(T) / sizeof(unsigned int) - 1;
	unsigned int i, k;
	size_t nodes_count = net.state().size();
	size_t threads_count = 32;
	size_t blocks_count = nodes_count / threads_count + 1;
	if(nodes_count % threads_count == 0) {
		--blocks_count;
	}
	
	thrust::device_vector<int> state0 = net.state(), xs = net.state(), ys = net.state();
	thrust::device_vector<node_behavior> dev_behavior = net.behavior();
	thrust::device_vector<int> functions_storage = net.functions();
	boolean_functions bfs(net.function_size(), net.arguments_size(), functions_storage);
	
	bool host_eq_to_ref;
	for(i = 1, k = 0; i < T[max]; ++i){
		int* xs_ptr = thrust::raw_pointer_cast(xs.data());
		int* ys_ptr = thrust::raw_pointer_cast(ys.data());
		node_behavior* b_ptr = thrust::raw_pointer_cast(dev_behavior.data());
		int* ref_ptr = thrust::raw_pointer_cast(state0.data());
		//int* ref_eq_ptr = thrust::raw_pointer_cast(equal_to_reference_state.data());
		
		host_eq_to_ref = false;
		hipMemcpyToSymbol(HIP_SYMBOL("dev_eq_to_ref"), &host_eq_to_ref, sizeof(bool), 0, hipMemcpyHostToDevice);
		//hipMemset(dev_eq_to_ref, true, sizeof(bool));

		update_state<<<blocks_count, threads_count>>> (
			nodes_count,
			xs_ptr,
			ys_ptr,
			b_ptr,
			bfs,
			ref_ptr//, // reference state
			//dev_eq_to_ref
		);

		xs.swap(ys);
	
		hipMemcpyFromSymbol(&host_eq_to_ref, HIP_SYMBOL("dev_eq_to_ref"), sizeof(bool), 0, hipMemcpyDeviceToHost);

		/*if(host_eq_to_ref) {
			break;
		}

		if(i == T[k]){
			++k;
			//cout << "ts";
			state0 = xs;
			thrust::fill(dev_behavior.begin(), dev_behavior.end(), node_behavior());
		}*/
	}
	
	thrust::host_vector<int> state1 = xs;
	net.state() = std::vector<int>(state1.begin(), state1.end());
	
	thrust::host_vector<node_behavior> behavior = dev_behavior;
	net.behavior() = std::vector<node_behavior>(behavior.begin(), behavior.end());
	
	if(i == T[max]){
		std::cout << "koniec";// << std::endl;
		return i - T[max - 1];
	}
	else if(k > 0) {
		return i - T[k-1];
	}
	else return i;
}
	
} // namespace gpu_acc
