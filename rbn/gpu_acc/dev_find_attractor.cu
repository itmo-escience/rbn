#include "hip/hip_runtime.h"
// Compiled with CUDA C++

#include "dev_find_attractor.hpp"

#include <iostream>
#include <thrust/functional.h>
#include <thrust/logical.h>
#include <thrust/fill.h>
#include <thrust/device_vector.h>
#include <thrust/swap.h>

#include "boolean_functions.hpp"

namespace gpu_acc {

namespace {

struct dev_rm {
	~dev_rm() {
		hipDeviceReset();
	}
};

dev_rm& dev_init() {
	static dev_rm drm;
	return drm;
}

__global__
void update_state(int iter, int nodes_count, const int* xs, int* ys, node_behavior* behavior,
					boolean_functions fs, int* ref, bool* already_found, bool* eq_to_ref)
{
	if(*already_found) return;
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if(idx < nodes_count) {
		int x = xs[idx];
		int y = fs(idx, xs);
		if(x != y) {
			behavior[idx].changes += 1;
		}
		behavior[idx].sum += y;
		ys[idx] = y;
		if(y != ref[idx]) {
			*eq_to_ref = false;
		}
	}
}

int equal_to_reference_on_iteration(const thrust::device_vector<bool>& etr_vector, int relative_to) {
	thrust::device_vector<bool>::const_iterator etr_on = thrust::find(etr_vector.begin(), etr_vector.end(), true);
	if(etr_on != etr_vector.end()) {
		return relative_to + thrust::distance(etr_vector.begin(), etr_on);
	} else {
		return 0;
	}
}

void clear_eq_to_ref(thrust::device_vector<bool>& etr_vector) {
	thrust::fill(etr_vector.begin(), etr_vector.end(), true);
	etr_vector[0] = false;
}

void clear_behavior(thrust::device_vector<node_behavior>& behavior) {
	thrust::fill(behavior.begin(), behavior.end(), node_behavior());
}

void get_results_from_gpu(thrust::device_vector<int>& state, thrust::device_vector<node_behavior>& behavior, network& net) {
	thrust::host_vector<int> h_state = state;
	net.state() = std::vector<int>(h_state.begin(), h_state.end());
	
	thrust::host_vector<node_behavior> h_behavior = behavior;
	net.behavior() = std::vector<node_behavior>(h_behavior.begin(), h_behavior.end());
}

} // namespace

int dev_find_attractor(network& net) {
	dev_init();
	unsigned int T[] = {100, 1000, 10000, 100000, 1000000};
	const int max = sizeof(T) / sizeof(unsigned int) - 1;
	unsigned int i, k;
	size_t nodes_count = net.state().size();
	size_t threads_count = 512;
	size_t blocks_count = nodes_count / threads_count + 1;
	if(nodes_count % threads_count == 0) {
		--blocks_count;
	}
	
	thrust::device_vector<int> state0 = net.state(), xs = net.state(), ys(net.state().size(), 0);
	thrust::device_vector<node_behavior> dev_behavior = net.behavior();
	thrust::device_vector<int> functions_storage = net.functions();
	boolean_functions bfs(net.function_size(), net.arguments_size(), functions_storage);
	
	int attractor_found_on = 0;
	// Synchronizing CPU and GPU after each kernel invokation (update_state) is very expensive
	// So we run (check_state_each) iterations, and then check if attractor was found
	// check_state_each must be less than T[k] for all k (will fix later)
	int check_state_each = 90;
	thrust::device_vector<bool> dev_eq_to_ref(check_state_each + 1);
	clear_eq_to_ref(dev_eq_to_ref);

	for(i = 1, k = 0; i < T[max]; ++i){
		int* xs_ptr = thrust::raw_pointer_cast(xs.data());
		int* ys_ptr = thrust::raw_pointer_cast(ys.data());
		node_behavior* b_ptr = thrust::raw_pointer_cast(dev_behavior.data());
		int* ref_ptr = thrust::raw_pointer_cast(state0.data());
		bool* eq_to_ref_ptr = thrust::raw_pointer_cast(dev_eq_to_ref.data());
		
		update_state<<<blocks_count, threads_count>>> (i,
			nodes_count,
			xs_ptr,
			ys_ptr,
			b_ptr,
			bfs,
			ref_ptr, // reference state
			eq_to_ref_ptr + ((i - 1) % check_state_each),
			eq_to_ref_ptr + ((i - 1) % check_state_each + 1)
		);

		xs.swap(ys);

		if(i % check_state_each == 0) {
			int relative_to = ((i - 1) / check_state_each) * check_state_each;
			attractor_found_on = equal_to_reference_on_iteration(dev_eq_to_ref, relative_to);
			if(attractor_found_on > 0) {
				break;
			}
			clear_eq_to_ref(dev_eq_to_ref);
			hipDeviceSynchronize();
		}

		if(i == T[k]){
			++k;
			state0 = xs;
			clear_behavior(dev_behavior);
		}
	}

	get_results_from_gpu(xs, dev_behavior, net);

	if(attractor_found_on == 0){
		std::cout << "koniec";// << std::endl;
		return T[max] - T[max - 1];
	}
	else if(k > 0) {
		return attractor_found_on - T[k-1];
	}
	else return attractor_found_on;
}
	
} // namespace gpu_acc
