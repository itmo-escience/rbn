#include "boolean_functions.hpp"

#include <thrust/device_vector.h>
#include <thrust/transform.h>
#include <thrust/functional.h>
#include <thrust/iterator/counting_iterator.h>

#include "../host/boolean_functions.hpp"
#include "state.hpp"

namespace gpu {

namespace detail {

namespace device {

boolean_functions::boolean_functions(const host::boolean_functions& fs)
        : m_storage(fs.data()), m_max_kin(fs.max_kin()) { }

} // namespace device

} // namespace detail

} // namespace gpu
